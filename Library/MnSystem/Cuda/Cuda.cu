#include "Cuda.h"
#include <cstdio>
#include <utility>

#include <cuda_occupancy.h>    ///<	for optimal kernel launching
#include <hip/hip_runtime_api.h> ///<	for evaluating kernel performance
#include <hip/hip_runtime.h>

#define MEM_POOL_CTRL 3

namespace mn {

KernelConfig::KernelConfig(KernelFunc f, 
                           hipFuncCache_t cacheConfig, // Shared mem usually bottleneck for ClaymoreUW. - Justin
                           bool isWave)
    : func(f), cachePreference(cacheConfig), waveFashion(isWave) {
  hipFuncGetAttributes(&attribs, reinterpret_cast<const void*>(f));
  maxOccBlockSize = Cuda::evalOptimalBlockSize(attribs, cachePreference);
  if (cacheConfig != hipFuncCachePreferNone) ///< should be different from
    ///< device cache preference
    checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(f), cacheConfig));
    checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(f), hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxShared)); // Should rebind for run-time / user setting that isn't Max Shared - Justin
}

Cuda::Cuda() : _default_devId{0} {
  printf("[Init -- Begin] Cuda\n");
  //< acquire devices
  _dev_num_available = 0;
  hipError_t error_id = hipGetDeviceCount(&_dev_num_available);
  if (error_id != hipSuccess) {
    printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id,
           hipGetErrorString(error_id));
    printf("Result = FAIL\n");
    exit(EXIT_FAILURE);
  }
  if (_dev_num_available == 0)
    printf("\t[InitInfo -- DevNum] There are no available device(s) that "
           "support CUDA\n");
  else
    printf("\t[InitInfo -- DevNum] Detected %d CUDA Capable device(s)\n",
           _dev_num_available);
  set_max_device(); //<[TMP]
  printf(
      "\t[InitInfo -- DevNum] Prepare to use %d device(s) in Multi-GPU test\n",
      _dev_num_using);

  _akDeviceProps.resize(_dev_num_available);
  _akStreams.resize(_dev_num_available);
  _akEvents.resize(_dev_num_available);

  for (int i = 0; i < _dev_num_available; i++) {
    // checkCudaErrors(hipSetDevice(i));
    checkCudaErrors(hipSetDevice(i));
    ///< device properties
    checkCudaErrors(hipGetDeviceProperties(&_akDeviceProps[i], i));
    const auto &prop{_akDeviceProps[i]};
    printf("\t[InitInfo -- Dev Property] GPU device %d (%d-th group on "
           "board)\n\t\tglobal memory: %llu bytes,\n\t\tshared memory per "
           "block: %llu bytes,\n\t\tregisters per SM: %d,\n\t\tMulti-Processor "
           "count: %d,\n\t\tSM compute capabilities: %d.%d.\n",
           i, prop.multiGpuBoardGroupID,
           (long long unsigned int)prop.totalGlobalMem,
           (long long unsigned int)prop.sharedMemPerBlock, prop.regsPerBlock,
           prop.multiProcessorCount, prop.major, prop.minor);

    ///< streams
    // int leastPriority = 0;
    // int greatestPriority = leastPriority;
    // hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);
    // hipStreamCreateWithPriority(&compute_stream, hipStreamDefault,
    // leastPriority); hipStreamCreateWithPriority(&push_top_stream,
    // hipStreamDefault, greatestPriority);
    // hipStreamCreateWithPriority(&push_bottom_stream, hipStreamDefault,
    // greatestPriority);
    // for (auto &stream : _akStreams[i])
    for (int j = 0; j < (int)StreamIndex::Total; ++j)
      checkCudaErrors(hipStreamCreate(&_akStreams[i][j]));
    printf("\t[InitInfo -- stream] Create %lu streams for device %d\n",
           (long unsigned)_akStreams[i].extent, i);

    // for (auto &event : _akEvents[i])
    for (int j = 0; j < (int)EventIndex::Total; ++j)
      checkCudaErrors(hipEventCreateWithFlags(
          &_akEvents[i][j], // hipEventDefault | hipEventBlockingSync |
          // hipEventInterprocess |
          hipEventDisableTiming));

    ///< memory allocator
    std::size_t free_byte, total_byte;
    checkCudaErrors(hipMemGetInfo(&free_byte, &total_byte));
    ///
    _akMonotonicAllocators.emplace_back(std::make_unique<MonotonicAllocator>(
        prop.textureAlignment,
        free_byte >> MEM_POOL_CTRL)); ///< preserve 1/4 space for intermediate
    ///< computations
    hipDeviceSynchronize();
    printf("\t[InitInfo -- memory] device %d\n\t\tfree bytes/total bytes: "
           "%lu/%lu,\n\t\tpre-allocated size: %lu bytes\n\n",
           i, (long unsigned)free_byte, (long unsigned)total_byte,
           (long unsigned)(free_byte >> MEM_POOL_CTRL));
  }

  //< enable peer access
  for (int i = 0; i < _dev_num_available; i++) {
    for (int j = 0; j < _dev_num_available; j++) {
      if (i != j) {
        establishPeerAccess(i, j);
        printf("\t[InitInfo -- Peer Access] Enable peer access from %d to %d\n",
               i, j);
      }
    }
  }
  //< init cuda context
  for (int i = 0; i < _dev_num_available; i++) {
    _akCuDev_contexts.emplace_back(i); //< set device when construct
  }

  printf("\t[InitInfo -- Default Dev] Default context: %d\n", _default_devId);
  checkCudaErrors(hipSetDevice(_default_devId));
  printf("\n[Init -- End] == Finished \'Cuda\' initialization\n\n");
  // getchar();
}

Cuda::~Cuda() {
  // hipStreamDestroy(_kMemCopyStream);
  getInstance()->_akDeviceProps.clear();
  for (auto &streams : getInstance()->_akStreams)
    // for (auto &stream : streams)
    for (int j = 0; j < (int)StreamIndex::Total; ++j)
      checkCudaErrors(hipStreamDestroy(streams[j]));
  for (auto &events : getInstance()->_akEvents)
    // for (auto &event : events)
    for (int j = 0; j < (int)EventIndex::Total; ++j)
      checkCudaErrors(hipEventDestroy(events[j]));
#if 0
  for (auto &monoAllocator : _akMonotonicAllocators)
    for (int i = 0; i < getInstance()->_akMonotonicAllocators.size(); ++i)
      getInstance()->_akMonotonicAllocators[i].~MonotonicAllocator();
#endif
  printf("  Finished \'Cuda\' termination\n");
}

int Cuda::generalGridSize(int &threadNum, int &blockSize) const {
  return (threadNum + blockSize - 1) / blockSize;
}
int Cuda::waveGridSize(int &threadNum, int &blockSize) const {
  auto blocksPerSM =
      (threadNum / blockSize /
       getInstance()->_akDeviceProps[_iDevID].multiProcessorCount) *
      getInstance()->_akDeviceProps[_iDevID].multiProcessorCount;
  return blocksPerSM ? blocksPerSM : 1;
}

/// static methods
int Cuda::evalOptimalBlockSize(hipFuncAttributes attribs,
                               hipFuncCache_t cachePreference, // Shared mem usually bottleneck for ClaymoreUW. - Justin
                               std::size_t smemBytes) {
  auto instance = getInstance();
  cudaOccDeviceProp prop =
      getInstance()->_akDeviceProps[instance->_iDevID]; ///< cache preference
  cudaOccFuncAttributes occAttribs = attribs;
  cudaOccDeviceState occCache;
  switch (cachePreference) {
  case hipFuncCachePreferNone:
    occCache.carveoutConfig = SHAREDMEM_CARVEOUT_DEFAULT; // Better to use carveoutConfig over cacheConfig for modern CUDA, - Justin
    occCache.cacheConfig = CACHE_PREFER_NONE;
    break;
  case hipFuncCachePreferShared:
    occCache.carveoutConfig = SHAREDMEM_CARVEOUT_MAX_SHARED; // Better to use carveoutConfig over cacheConfig for modern CUDA, - Justin
    occCache.cacheConfig = CACHE_PREFER_SHARED;
    break;
  case hipFuncCachePreferL1:
    occCache.carveoutConfig = SHAREDMEM_CARVEOUT_MAX_L1; // Better to use carveoutConfig over cacheConfig for modern CUDA, - Justin
    occCache.cacheConfig = CACHE_PREFER_L1;
    break;
  case hipFuncCachePreferEqual:
    occCache.carveoutConfig = SHAREDMEM_CARVEOUT_HALF; // Better to use carveoutConfig over cacheConfig for modern CUDA, - Justin
    occCache.cacheConfig = CACHE_PREFER_EQUAL;
    break;
  default:; ///< should throw error
  }
  int minGridSize, blockSize = 32;
  cudaOccMaxPotentialOccupancyBlockSize(&minGridSize, &blockSize, &prop,
                                        &occAttribs, &occCache, nullptr,
                                        smemBytes);
  return blockSize;
}

ExecutionPolicy Cuda::launchConfig(std::string kernelName, int threadNum,
                                   bool sync, std::size_t smemSize,
                                   hipStream_t sid) const {
  auto instance = getInstance();
  if (instance->_kFuncTable.find(kernelName) == instance->_kFuncTable.end()) {
    int bs = 256;
    printf("Warning: Kernel function %s not registered! Use 256 setting!\n",
           kernelName.data());
    return {generalGridSize(threadNum, bs), bs, smemSize, sync};
  }
  auto &config = instance->_kFuncTable[kernelName.data()];
  int bs = config.maxOccBlockSize;
  if (smemSize > 0)
    bs = evalOptimalBlockSize(config.attribs, config.cachePreference, smemSize);
  // printf("configurating for kernel[%s] blocksize: %d\n", kernelName.c_str(),
  // bs);
  if (config.waveFashion)
    return {waveGridSize(threadNum, bs), bs, smemSize, sync};
  return {generalGridSize(threadNum, bs), bs, smemSize, sync};
}

void Cuda::registerKernel(std::string tag, KernelFunc f,
                          hipFuncCache_t cacheConfig, // Shared mem usually bottleneck for ClaymoreUW. - Justin, 
                          bool waveFashion) {
  auto instance = getInstance();
  instance->_kFuncTable.emplace(tag, KernelConfig(f, cacheConfig, waveFashion));
  printf("Kernel[%s](%s) block size configuration: %d\n", tag.data(),
         waveFashion ? "wave" : "general",
         instance->_kFuncTable[tag.data()].maxOccBlockSize);
}
const KernelConfig &Cuda::findKernel(std::string tag) {
  auto instance = getInstance();
  return instance->_kFuncTable[tag.data()];
}

} // namespace mn
